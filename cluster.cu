#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <fcntl.h>
#include <stdlib.h>
#include <math.h>

#include <string>
#include <iostream>
#include <fstream>
#include <vector>
#include <sstream>
#include <algorithm>
#include <utility>
#include <set>
#include <chrono>
#include <iomanip>

//thrust utils (for GPU accel)

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/copy.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>

using namespace std;
using namespace std::chrono;

#include "dataread.h"

struct range {
  float min;
  float max;
};

struct edgedata {
  float weight;
  int vertex;
};

struct location {
  float x;
  float y;
};

#include "cluster_device.cuh"

//device error check
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//floating point comps
bool cmfl(float x, float y, float epsilon = 0.01f){
  if(fabs(x - y) < epsilon)
     return true; //they are same
     return false; //they are not same
}

//get index of edge from 1d representation of a matrix
int getMatrixIndex(int record_size, int i, int j){
    return (i * record_size) + j;
}

void printRecords(vector<vector<float> > records){

    int i, j;
    for(i = 0; i < records.size(); i++){
        for(j = 0; j < records[i].size(); j++){
            cout << records[i][j] << ' ';
        }
        cout << endl;
    }

}

bool symbolic(int index){
  if(index == 1 || index == 2 || index == 3){
    return true;
  }
  return false;
}

/*
get range [min, max] of all continous attributes
*/
vector<range> attributeRanges(vector<vector<float> > records){

  vector<range> ranges;

  int i, j;
  int attr_count = records[0].size();
  for(i = 0; i < attr_count; i++){

    int attr = i;

    //init min, max
    int min = records[0][attr];
    int max = records[0][attr];

    for(j = 0; j < records.size(); j++){

      //update min/max
      float curr = records[j][attr];
      if(curr < min) min = curr;
      if(curr > max) max = curr;

    }

    //save min/max in ranges
    range curr_range;
    curr_range.min = min;
    curr_range.max = max;
    ranges.push_back(curr_range);
  }

  return ranges;

}

float scaleValue(float r_min, float r_max, float t_min, float t_max, float m){

  //if rmax == rmin (usualy for 0 values) return t_min (usually 0, to symbolize all values will be identical)
  if(r_max == r_min){
    return t_min;
  }

  float val = (((m - r_min) / (r_max - r_min)) * (t_max - t_min) + t_min);

  return val;

}

void vectorsubtraction(vector<float> v1, vector<float> v2, vector<float> &v3){

    int i;
    //vectors must be same size
    if(v1.size() != v2.size()) return;
    for(i = 0; i < v1.size(); i++){
        //if symbolic attribute, push 1 to represent not equal, 0 to represent equal
        if(symbolic(i)){
          v3.push_back(v1[i] != v2[i]);
          //if continous, subtract values
        } else {
          v3.push_back((v1[i] - v2[i]));
        }
    }
}

float normalizevector(vector<float> v){

    int i;
    float total = 0.0;
    for(i = 0; i < v.size(); i++){
        total += (v[i] * v[i]);
    }

    return sqrt(total);
}

double edgeweight(vector<float> v1, vector<float> v2){

    vector<float> v3;
    vectorsubtraction(v1, v2, v3);

    float normalized = normalizevector(v3);
    double base = exp(normalized);
    return (1 / base);
}

int main(){

    /*
    init dataset
    */

    vector<vector<float> > records;
    vector<int> record_types;
    int record_count, record_size;
    init_dataset(record_count, record_size, records, record_types);

    cout <<  record_count << endl;


    //double edge = edgeweight(records[0], records[4]);

    vector<range> ranges = attributeRanges(records);

    int i, j;

    for(i = 0; i < ranges.size(); i++){
      cout << "(" << ranges[i].min << ", " << ranges[i].max << ") ";
    }

    cout << endl;

    /*
    scale values
    */

    for(i = 0; i < records.size(); i++){
      for(j = 0; j < records[i].size(); j++){
        if(symbolic(j)){ //unless symbolic attribute
          continue;
        }
        records[i][j] = scaleValue(ranges[j].min, ranges[j].max, 0, 1, records[i][j]);
      }
    }

    /*
    populate edge matrix
    */

    //printRecords(records);

    int k = 5;

    float *recordmatrix;
    edgedata *edgematrix;
    int edge_msize = records.size() * k;//records.size();
    int records_msize = records.size() * record_size;
    cout << "ems: " << edge_msize << ", " << records_msize << endl;
    gpuErrchk(hipMallocManaged(&edgematrix, edge_msize * sizeof(edgedata)));
    gpuErrchk(hipMallocManaged(&recordmatrix, records_msize * sizeof(float)));

    //copy records to a unified array from stl vector (so it can work on cuda kernel)
    int curr = 0;
    for(i = 0; i < records.size(); i++){
      for(j = 0; j < record_size; j++){
        recordmatrix[curr] = records[i][j];
        curr++;
      }
    }

    //init all elements in edge matrix to -1
    for(i = 0; i < edge_msize; i++){
      edgematrix[i].weight = -1.0;
    }

    /*
    printRecords(records);
    cout << "-------------\n";
    for(i = 0; i < record_count; i++){
      for(j = 0; j < record_size; j++){
        int currindex = getMatrixIndex(record_size, i, j);
        cout << recordmatrix[currindex] << " ";
      }
      cout << endl;
    }
    */

    //start kernel
 
  
    edgeGeneration<<<16, 64>>>(edgematrix, recordmatrix, record_size, record_count, k);
    hipDeviceSynchronize();

    cout << "--------------\n";

    //printRecords(records);


    
    //print k nearest neighbors
    for(i = 0; i < 50; i++){
      for(j = 0; j < k; j++){
        int curredge = getMatrixIndex(k, i, j);
        cout << setprecision(2) << edgematrix[curredge].weight << " ";
        //cout << setprecision(4) << "( " << edgematrix[curredge].weight << " " << edgematrix[curredge].vertex << ") ";
      }
      cout << endl;
    }

    cout << "........\n";

    for(i = record_count-10; i < record_count; i++){
      for(j = 0; j < k; j++){
        int curredge = getMatrixIndex(k, i, j);
        cout << setprecision(2) << edgematrix[curredge].weight << " ";
        //cout << setprecision(4) << "( " << edgematrix[curredge].weight << " " << edgematrix[curredge].vertex << ") ";
      }
      cout << endl;
    }
    

    /*
    begin clustering approach
    */

  
    /*
    generate random positions based on indices of records
    */

    int *p = new int[record_count];
    for(i = 0; i < record_count; i++){
      //init locations to its source vertex index
      p[i] = i;
    }

    for(i = 0; i < record_count; i++){
      //assign each vertex a random location
      p[i] = rand() % (record_count);
    }

    for(i = 0; i < 50; i++){
      cout << p[i] << endl;
    }

    int rounds = 15;
    int l;
    for(l = 0; l < rounds; l++){

      for(i = 0; i < record_count; i++){

        //move vertices based on force from neighbors
        float distance = 0.0;
        for(j = 0; j < k; j++){
  
          //get vertex to compare to
          int compedge = getMatrixIndex(k, i, j);
          int compvertex = edgematrix[compedge].vertex;
 
          if(edgematrix[compedge].weight == -1) continue;

          //move distance between two vertices proportional to weight between them
          p[i] -= ((p[i] - p[compvertex]) * edgematrix[compedge].weight);
        }
  
        //p[i] += distance;
      }
  
      //remove edges with lower than average scores
      for(i = 0; i < record_count; i++){
  
        float total = 0.0;
        for(j = 0; j < k; j++){
          int curredge = getMatrixIndex(k, i, j);
  
          //skip if edge has been cut
          if(edgematrix[curredge].weight == -1) continue;
  
          total += edgematrix[curredge].weight;
        }
  
        float avg = (total / k);
  
        for(j = 0; j < k; j++){
          int curredge = getMatrixIndex(k, i, j);
          if(edgematrix[curredge].weight < avg){
            edgematrix[curredge].weight = -1;
          }
        }
      }

    }

    cout << "------------------------\n";

    for(i = 0; i < 10; i++){
      cout << p[i] << endl;
    }
    cout << "~~~~~~~~~~~\n";
    for(i = record_count-10; i < record_count; i++){
      cout << p[i] << endl;
    }


    /*
    int l;
    for(l = 0; l < 5; l++){

      for(i = 0; i < record_count; i++){

        float total = 0.0;
        for(j = 0; j < k; j++){
          int curredge = getMatrixIndex(k, i, j);

          //skip if edge has been cut
          if(edgematrix[curredge].weight == -1) continue;

          total += edgematrix[curredge].weight;
        }
  
        float avg = (total / k);
  
        if(i > 5490){
          cout << "avg: " << avg << ", " << total << endl;
        }
        for(j = 0; j < k; j++){
          int curredge = getMatrixIndex(k, i, j);
          if(edgematrix[curredge].weight < avg){
            edgematrix[curredge].weight = -1;
          }
        }
      }
    }
  
    for(i = 0; i < 50; i++){
      for(j = 0; j < k; j++){
        int curredge = getMatrixIndex(k, i, j);
        cout << setprecision(2) << edgematrix[curredge].weight << " ";
        //cout << setprecision(4) << "( " << edgematrix[curredge].weight << " " << edgematrix[curredge].vertex << ") ";
      }
      cout << endl;
    }

    cout << "........\n";

    for(i = record_count-10; i < record_count; i++){
      for(j = 0; j < k; j++){
        int curredge = getMatrixIndex(k, i, j);
        cout << setprecision(2) << edgematrix[curredge].weight << " ";
        //cout << setprecision(4) << "( " << edgematrix[curredge].weight << " " << edgematrix[curredge].vertex << ") ";
      }
      cout << endl;
    }
*/

/*
    int i, j;
    for(i = 0; i < records.size(); i++){
        for(j = 0; j < records.size(); j++){
            if(i == j) cout << i << ", " << j << ": " << 0 << endl;
            cout << i << ", " << j << ": " << edgeweight(records[i], records[j]) << endl;
        }
    }
*/
    return 0;
}
